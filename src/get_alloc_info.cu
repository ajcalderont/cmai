
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void emptyKernel()
{
}

#define cudaCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      printf("CUDA error: %s - %s(%d)\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int main(int argc, char* argv[])
{
	// Initialization
	/*----------------------------------------------------------------------------------------*/
	int device = atoi(argv[1]);
	cudaCheck(hipSetDevice(device));
	hipSetDeviceFlags(hipDeviceMapHost);

	int runtime_version;
	int driver_version;
	hipDeviceProp_t properties;
	hipGetDeviceProperties(&properties, device);
	hipRuntimeGetVersion(&runtime_version);
	hipDriverGetVersion(&driver_version);

	emptyKernel<<<1,1>>>();
	hipDeviceSynchronize();
	
	// Pool size
	/*----------------------------------------------------------------------------------------*/
	int pool_size = 1;
	char *h_data;
	break_01: __attribute__((unused));
	hipHostAlloc((void**)&h_data, 1, hipHostMallocMapped);
	hipHostFree(h_data);
	
	// Maximum allocations and granularity
	/*----------------------------------------------------------------------------------------*/
	char **h_data_array = (char**) malloc(pool_size * sizeof(char*));
	hipHostAlloc((void**)&h_data_array[0], 1, hipHostMallocMapped);
	break_02: __attribute__((unused));
	int granularity = 0, iteration = 0, flag = 0;
	while(!flag && iteration < pool_size)
	{
		iteration++; 
		hipHostAlloc((void**)&h_data_array[iteration], 1, hipHostMallocMapped);
	}
	for(int i = 0; i <= iteration; i++)
	{
		hipHostFree(h_data_array[i]);
	}
	free(h_data_array);
	
	// Size classes
	/*----------------------------------------------------------------------------------------*/
	char *h_data_inf, *h_data_sup;
	int inf_size = granularity, sup_size = granularity, finished = 1, class_finished = 0;
	break_03: __attribute__((unused));
	hipHostAlloc((void**)&h_data_inf, inf_size, hipHostMallocMapped);
	while(!finished)
	{
		sup_size = sup_size + granularity;
		hipHostAlloc((void**)&h_data_sup, sup_size, hipHostMallocMapped);
		hipHostFree(h_data_sup);
		if(class_finished)
		{
			class_finished = 0;
			hipHostFree(h_data_inf);
			inf_size = sup_size;
			hipHostAlloc((void**)&h_data_inf, inf_size, hipHostMallocMapped);
		}
	}
	hipHostFree(h_data_inf);
	
	// Larger allocations
	/*----------------------------------------------------------------------------------------*/
	break_04: __attribute__((unused));
	hipHostAlloc((void**)&h_data, pool_size + 1, hipHostMallocMapped);
	hipHostFree(h_data);

	// Allocator policy
	/*----------------------------------------------------------------------------------------*/
	char *chunk_1, *chunk_2, *chunk_3, *chunk_4, *chunk_5, *chunk_6, *chunk_7, *chunk_8, *chunk_9, *chunk_10;
	hipHostAlloc((void**)&chunk_1, granularity * 2, hipHostMallocMapped);
	hipHostAlloc((void**)&chunk_2, granularity, hipHostMallocMapped);
	hipHostAlloc((void**)&chunk_3, granularity * 2, hipHostMallocMapped);
	hipHostAlloc((void**)&chunk_4, granularity, hipHostMallocMapped);
	hipHostAlloc((void**)&chunk_5, granularity, hipHostMallocMapped);
	hipHostAlloc((void**)&chunk_6, granularity, hipHostMallocMapped);
	hipHostFree(chunk_1);
	hipHostFree(chunk_3);
	hipHostFree(chunk_5);
	hipHostAlloc((void**)&chunk_7, granularity, hipHostMallocMapped);
	hipHostAlloc((void**)&chunk_8, granularity, hipHostMallocMapped);
	break_05: __attribute__((unused));
	hipHostFree(chunk_2);
	hipHostFree(chunk_4);
	hipHostFree(chunk_6);
	hipHostFree(chunk_7);
	hipHostFree(chunk_8);

	// Coalescing support
	/*----------------------------------------------------------------------------------------*/
	hipHostAlloc((void**)&chunk_1, granularity, hipHostMallocMapped);
	hipHostAlloc((void**)&chunk_2, granularity, hipHostMallocMapped);
	hipHostAlloc((void**)&chunk_3, granularity, hipHostMallocMapped);
	hipHostFree(chunk_1);
	hipHostFree(chunk_2);
	hipHostAlloc((void**)&chunk_4, granularity * 2, hipHostMallocMapped);
	break_06: __attribute__((unused));
	hipHostFree(chunk_3);
	hipHostFree(chunk_4);

	// Splitting support
	/*----------------------------------------------------------------------------------------*/
	hipHostAlloc((void**)&chunk_1, granularity * 2, hipHostMallocMapped);
	hipHostAlloc((void**)&chunk_2, granularity, hipHostMallocMapped);
	hipHostFree(chunk_1);
	hipHostAlloc((void**)&chunk_3, granularity, hipHostMallocMapped);
	break_07: __attribute__((unused));
	hipHostFree(chunk_2);
	hipHostFree(chunk_3);

	// Expansion policy
	/*----------------------------------------------------------------------------------------*/
	int max_allocations = pool_size / granularity;
	h_data_array = (char**) malloc(max_allocations * sizeof(char*));
	hipHostAlloc((void**)&h_data_array[0], granularity, hipHostMallocMapped);
	break_08: __attribute__((unused));
	int index;
	for(index = 1; index < max_allocations; index++)
	{
		hipHostAlloc((void**)&h_data_array[index], granularity, hipHostMallocMapped);
	}
	for(index = 0; index < max_allocations; index++)
	{
		hipHostFree(h_data_array[index]);
	}
	free(h_data_array);


	// Pool usage
	/*----------------------------------------------------------------------------------------*/
	int quarter = pool_size / 4;
	hipHostAlloc((void**)&chunk_1, quarter, hipHostMallocMapped);
	hipHostAlloc((void**)&chunk_2, quarter, hipHostMallocMapped);
	hipHostAlloc((void**)&chunk_3, quarter, hipHostMallocMapped);
	hipHostAlloc((void**)&chunk_4, quarter, hipHostMallocMapped);
	hipHostAlloc((void**)&chunk_5, quarter, hipHostMallocMapped);
	hipHostAlloc((void**)&chunk_6, quarter, hipHostMallocMapped);
	hipHostAlloc((void**)&chunk_7, quarter, hipHostMallocMapped);
	hipHostAlloc((void**)&chunk_8, quarter, hipHostMallocMapped);
	hipHostAlloc((void**)&chunk_9, quarter, hipHostMallocMapped);
	hipHostFree(chunk_1);
	hipHostFree(chunk_2);
	hipHostFree(chunk_5);
	hipHostAlloc((void**)&chunk_10, quarter, hipHostMallocMapped);
	break_09: __attribute__((unused));
	hipHostFree(chunk_10);

	// Shrinking support
	/*----------------------------------------------------------------------------------------*/
	flag = 0;
	break_10: __attribute__((unused));
	hipHostFree(chunk_6);
	hipHostFree(chunk_7);
	hipHostFree(chunk_8);
	flag = 1;
	hipHostFree(chunk_9);
	flag = 2;
	hipHostFree(chunk_3);
	hipHostFree(chunk_4);

	// Finalization
	/*----------------------------------------------------------------------------------------*/
	hipDeviceReset();
	return 0;
}